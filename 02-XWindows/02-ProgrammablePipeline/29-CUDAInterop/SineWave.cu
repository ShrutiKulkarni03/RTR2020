
#include <hip/hip_runtime.h>
//CUDA KERNEL DEFINITION
//global kernel function declaration
__global__ void SineWave_kernel(float4* pos, unsigned int meshWidth, unsigned int meshHeight, float time)
{
	//variable declarations
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	float u = x / (float)meshWidth;
	float v = y / (float)meshHeight;

	u = u * 2.0f - 1.0f;
	v = v * 2.0f - 1.0f;

	float frequency = 4.0f;

	float w = sinf(u * frequency + time) * cosf(v * frequency + time) * 0.5f;

	pos[y * meshWidth + x] = make_float4(u, w, v, 1.0f);

}

void launchCUDAKernel(float4* pos, unsigned int meshWidth, unsigned int meshHeight, float time)
{
	dim3 block(8, 8, 1);
	dim3 grid(meshWidth / block.x, meshHeight / block.y, 1);

	/*dim3 block = dim3(8, 8, 1);
	dim3 grid = dim3(meshWidth / block.x, meshHeight / block.y, 1);*/
	

	SineWave_kernel <<< grid, block >>>
		(pos, meshWidth, meshHeight, time);
}

